#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/pushin_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

		template <typename Dtype>
		__global__ void PushinForward(const int n, const Dtype* in,
																	unsigned int* mask, const unsigned int threshold, const float scale,
																	Dtype* out, int st_count) {
						for (int i = 0; i < st_count; ++i)
								mask[i] = 1;
						for (int i = st_count; i < n; ++i)
								mask[i] = 0;

				CUDA_KERNEL_LOOP(index, n) {
						out[index] = in[index] * (mask[index] > threshold) * scale;
				}
		}

		template <typename Dtype>
		void PushinLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
														 						 const vector<Blob<Dtype>*>& top) {
				const Dtype* bottom_data = bottom[0]->gpu_data();
				Dtype* top_data = top[0]->mutable_gpu_data();
				const int count = bottom[0]->count();
				//if (this->phase_ == TRAIN) {
						unsigned int* mask =
								static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());
						//caffe_gpu_rng_uniform(count, mask);
						// set thresholds
						// NOLINT_NEXT_LINE(whitespace/operators)
						
						
						PushinForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
										count, bottom_data, mask, uint_thres_, scale_, top_data, st_count);
						CUDA_POST_KERNEL_CHECK;
				//} else {
				//		caffe_copy(count, bottom_data, top_data);
				//}
						st_count += 20;

						if(st_count > count)
								st_count = count;
		}

		template <typename Dtype>
		__global__ void PushinBackward(const int n, const Dtype* in_diff,
																	 unsigned int* mask, const unsigned int threshold, const float scale,
																	 Dtype* out_diff, int st_count) {
								for (int i = 0; i < st_count; ++i)
										mask[i] = 1;
								for (int i = st_count; i < n; ++i)
										mask[i] = 0;

				CUDA_KERNEL_LOOP(index, n) {
						out_diff[index] = in_diff[index] * scale * (mask[index] > threshold);
				}
		}

		template <typename Dtype>
		void PushinLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
																					const vector<bool>& propagate_down,
																					const vector<Blob<Dtype>*>& bottom) {
				if (propagate_down[0]) {
						const Dtype* top_diff = top[0]->gpu_diff();
						Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
						//if (this->phase_ == TRAIN) {
								unsigned int* mask =
										static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());
								//const unsigned int* mask =
								//		static_cast<const unsigned int*>(rand_vec_.gpu_data());
								const int count = bottom[0]->count();
								// NOLINT_NEXT_LINE(whitespace/operators)
								//
								//
								PushinBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
										CAFFE_CUDA_NUM_THREADS>>>(
														//count, top_diff, mask, uint_thres_, scale_, bottom_diff);
														count, top_diff, mask, uint_thres_, scale_, bottom_diff, st_count);
								CUDA_POST_KERNEL_CHECK;
						//} else {
						//		caffe_copy(top[0]->count(), top_diff, bottom_diff);
						//}
								st_count += 20;

								if(st_count > count)
										st_count = count;
				}
		}

		INSTANTIATE_LAYER_GPU_FUNCS(PushinLayer);

}  // namespace caffe
